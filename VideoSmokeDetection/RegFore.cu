#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cvpreload.h"
using namespace cv;
using namespace std;

__global__ void RegFore_Kenrel(uchar *fore, uchar *color_frame,int width, int height, uchar *reg_frame)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;


	if (x < width && y < height)
	{
		int offset = x + y * width;
		int r_offset = x * 3 + y * 3 * width;
		int g_offset = x * 3 + y * 3 * width + 1;
		int b_offset = x * 3 + y * 3 * width + 2;
		if (*(fore+offset)>0)
		{
			*(reg_frame + r_offset) = *(color_frame + r_offset);
			*(reg_frame + g_offset) = *(color_frame + g_offset);
			*(reg_frame + b_offset) = *(color_frame + b_offset);
		}
		else
		{
			*(reg_frame + r_offset) = 0;
			*(reg_frame + g_offset) = 0;
			*(reg_frame + b_offset) = 0;
		}

	}
}


extern "C" void RegFrame_Caller(const Mat &colorMat, const Mat &foreMask, int width, int height, Mat &RegMat)
{
	Mat resize_fore_mask;
	if (foreMask.size().width != width || foreMask.size().height != height)
	{
		resize(foreMask, resize_fore_mask, cvSize(width, height));
	}
	else resize_fore_mask = foreMask;

	if (RegMat.empty())
	{
		RegMat = Mat::zeros(cvSize(width, height), CV_8UC3);
	}

	size_t memSize = width*height*sizeof(uchar);

	uchar *d_fore_mat = NULL;
	uchar *d_color_mat = NULL;
	uchar *d_reg_mat = NULL;

	hipMalloc((void**)&d_fore_mat, memSize);
	hipMalloc((void**)&d_color_mat, 3*memSize);
	hipMalloc((void**)&d_reg_mat, 3 * memSize);

	hipError_t err;
	err = hipMemcpy(d_fore_mat, resize_fore_mask.data, memSize, hipMemcpyHostToDevice);
	err = hipMemcpy(d_color_mat, colorMat.data, 3*memSize, hipMemcpyHostToDevice);

	dim3 threads(32, 32);
	dim3 grids((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	RegFore_Kenrel << <grids, threads >> >(d_fore_mat, d_color_mat, width, height, d_reg_mat);

	err = hipMemcpy(RegMat.data, d_reg_mat, 3*memSize, hipMemcpyDeviceToHost);

	hipFree(d_fore_mat);
	hipFree(d_color_mat);
	hipFree(d_reg_mat);

}